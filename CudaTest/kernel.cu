#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */

#include <hip/hip_cooperative_groups.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h" // helper utility functions 

#include <opencv2/opencv.hpp>

#include <cstdio>
#include <cstdint>
#include <cstdlib>


using namespace cv;
namespace cg = cooperative_groups;

hipError_t lbpMultiscaleCuda(uint8_t* const c, const uint8_t* const a);

template<typename T>
__device__ inline T my_abs_dif(T a, T b) {
    return a > b ? (a - b) : (b - a);
}

__constant__ int8_t d[16]; 
__global__ void lbpKernel(uint8_t* const c, const uint8_t* const a, uint32_t w, uint32_t h, uint32_t r)
{

    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t i = blockIdx.y * blockDim.y + threadIdx.y;


    int32_t prev = -1;
    uint8_t U = 0;
    uint8_t D = 0;
    uint8_t M = 0;
    uint16_t temp = 0;

            uint16_t    pc   = 0;
    const   uint8_t     pa   = a[i * w + j];

    c[i * w + j] = pa;
   // return;

    if (i > r && j > r && i + r < h - 1 && j + r < w - 1) {

        for(uint8_t ii = 0; ii < 16; ii+=2){
            temp = my_abs_dif(pa, a[(i + d[ii]) * w + (j + d[ii + 1])]);
            
            if (M < temp) {
                D = 7-(ii>>1);
                M = temp;
            }
        }

        for (uint8_t ii = 0; ii < 16; ii+=2) {
            pc <<= 1;
            pc |= (a[(i + d[ii]) * w + (j + d[ii + 1])] > pa);

            if (prev > -1)
                U += (pc ^ prev) & 1;
            prev = pc;
        }

        
        if (U + (pc ^ (pc >> 7)) > 2) 
            c[i * w + j] = 9;
            
        else {
            temp = (pc >> D) | (pc << (8 - D));

            c[i * w + j] = temp;
        }
    }


}

void lbpKernelCpu(uint8_t* const c, const uint8_t* const a, uint32_t w, uint32_t h, uint32_t r) {

    int32_t prev = -1;
    uint8_t U = 0;
    uint8_t D = 0;
    uint8_t M = 0;
    uint16_t temp = 0;

    uint16_t    pc = 0;

    for(int i=0; i<h/2; i++)
        for (int j = 0; j < w/2; j++) {
            const   uint8_t     pa = *(a + (i * w + j));

            c[i * w + j] = 0;
            // return;

            if (i > r && j > r && i + r < h - 1 && j + r < w - 1) {

                for (uint8_t ii = 0; ii < 16; ii += 2) {
                    temp = abs(pa - a[(i + d[ii]) * w + (j + d[ii + 1])]) > M ? abs(pa - a[(i + d[ii]) * w + (j + d[ii + 1])]) : M;

                    if (M < temp) {
                        D = 7 - (ii >> 1);
                        M = temp;
                    }
                }

                for (uint8_t ii = 0; ii < 16; ii += 2) {
                    pc <<= 1;
                    pc |= (a[(i + d[ii]) * w + (j + d[ii + 1])] > pa);

                    if (prev > -1)
                        U += (pc ^ prev) & 1;
                    prev = pc;
                }


                if (U + (pc ^ (pc >> 7)) > 2)
                    c[i * w + j] = 9;

                else {
                    temp = (pc >> D) | (pc << (8 - D));

                    c[i * w + j] = temp;
                }
            }
        }
}


__global__ void histogramKernel(uint32_t H[256], uint8_t* a, uint32_t w) {
    uint32_t l_hist = 0;

    uint32_t j = threadIdx.x;
    uint32_t i = threadIdx.y;
    uint16_t pos = i * (w>>2) + j;


    uint32_t temp = ((uint32_t*)a)[pos + blockIdx.y * blockDim.y * blockDim.x];


#pragma unroll
    for (int i = 0; i < 4; i++) {
        l_hist <<= 8;
        l_hist |= (temp & 0xFF);
        temp >>= 8;
    }
  
#pragma unroll
    for (int i = 0; i < 4; i++) {
        atomicAdd(H + (l_hist & 0xFF), 1);
        l_hist >>= 8;
    }
}

void cpu_hist(uint32_t H[256], const uint8_t* const a, int w, int h) {

    for(int i=0; i<h; i++)
        for (int j = 0; j < w; j++) 
            H[a[i * w + j]]++;
}

hipError_t lbpMultiscaleCuda(uint8_t* const c, const uint8_t* const a)
{

    int8_t d_temp[16] = { 0, -1, 1, -1, 1, 0, 1, 1, 0, 1, -1, 1, -1, 0, -1, -1 };
    hipMemcpyToSymbol(HIP_SYMBOL(d), d_temp, 16);
    hipMemcpyFromSymbol(d_temp, HIP_SYMBOL(d), 16);

    uint8_t     w = 64,
        h = 64;
    uint16_t size = w * h;

    dim3 numThreads(16, 32);
    dim3 numBlocks1(4, 2);
    dim3 numBlocks2(1, 2);

    uint8_t* dev_a = 0;
    uint8_t* dev_c = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc(&dev_c, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_a, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }


    float gpu_time = 0.f;
    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface* timer = NULL;


    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);


    checkCudaErrors(hipDeviceSynchronize());

    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    
    uint32_t hist[256], *d_hist, hist2[256];
    hipMalloc(&d_hist, 256<<2);

    memset(hist , 0, sizeof(hist ));
    memset(hist2, 0, sizeof(hist2));
    hipMemset(d_hist, 0, sizeof(hist));

    lbpKernel <<< numBlocks1, numThreads >>> (dev_c, dev_a, w, h, 0);
    cudaStatus = hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    histogramKernel << <numBlocks2, numThreads >> > (d_hist, dev_c, w);
    checkCudaErrors(hipDeviceSynchronize());
    
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    

    
    hipMemcpy(hist, d_hist, 256 << 2, hipMemcpyDeviceToHost);
    
    hipFree(d_hist);


    //checkCudaErrors(hipDeviceSynchronize());
    

    hipEventElapsedTime(&gpu_time, start, stop);
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    // Launch a kernel on the GPU with one thread for each element.


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


Error:
    hipFree(dev_c);
    hipFree(dev_a);

    return cudaStatus;
}


hipError_t lbpMultiscaleCudaCpu(uint8_t* const c, const uint8_t* const a)
{

    int8_t d_temp[16] = { 0, -1, 1, -1, 1, 0, 1, 1, 0, 1, -1, 1, -1, 0, -1, -1 };
    hipMemcpyToSymbol(HIP_SYMBOL(d), d_temp, 16);
    hipMemcpyFromSymbol(d_temp, HIP_SYMBOL(d), 16);

    uint8_t     w = 64,
        h = 64;
    uint16_t size = w * h;

    dim3 numThreads(16, 32);
    dim3 numBlocks(4, 2);

    uint8_t* dev_a = 0;
    uint8_t* dev_c = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc(&dev_c, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_a, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipMalloc failed!");
        goto Error;
    }


    float gpu_time = 0.f;
    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface* timer = NULL;


    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);


    checkCudaErrors(hipDeviceSynchronize());

    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    //test <<<1, 256, 0, 0 >>> ();
    for (int i = 0; i < 10000; i++) {
        lbpKernelCpu(c, a, w, h, 0);
        //hipDeviceSynchronize();
    }
    //kernelTest(d1, c, a, w, h, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    //checkCudaErrors(hipDeviceSynchronize());


    hipEventElapsedTime(&gpu_time, start, stop);
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    // Launch a kernel on the GPU with one thread for each element.


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch

Error:
    hipFree(dev_c);
    hipFree(dev_a);

    return cudaStatus;
}



int main(int argc, char* argv[]) {
    Mat img = imread("C:\\users\\vlads\\desktop\\download.jfif", IMREAD_GRAYSCALE);
    resize(img, img, Size(64, 64), 0, 0);

    Mat dst(img.size(), CV_8U);

    uint8_t* const pimg = img.ptr();
    uint8_t* const pdst = dst.ptr();

    /*for (uint16_t i = 0; i < 16; i++)
    {
        lbpMultiscaleCuda(pdst, pimg);
    }*/


    lbpMultiscaleCuda(pdst, pimg);
    
    return 0;
}

int main1() {
    int8_t d[16] = { 0, -1, 1, -1, 1, 0, 1, 1, 0, 1, -1, 1, -1, 0, -1, -1 };
    int m[3][3] = {
                    {1, 2, 3},
                    {4, 5, 6},
                    {7, 8, 9}
                    };

    int i = 1, j = 1;
    uint8_t M=0, D=0, temp, U=0, w=64, pc=0;
     int8_t prev = -1;


    for (uint8_t ii = 0; ii < 16; ii += 2) {
        temp = abs(m[1][1] - m[1 + d[ii]][1 + d[ii + 1]]) > M ? abs(m[1][1] - m[1 + d[ii]][1 + d[ii + 1]]) : M;

        if (M < temp) {
            D = 7 - (ii >> 1);
            M = temp;
        }
    }

    for (uint8_t ii = 0; ii < 16; ii += 2) {
        pc <<= 1;
        pc |= (m[(i + d[ii])][(j + d[ii + 1])] > m[1][1]);

        if (prev > -1)
            U += (pc ^ prev) & 1;
        prev = pc;
    }


    temp = (pc >> D) | (pc << (8 - D));

    return 0;
}